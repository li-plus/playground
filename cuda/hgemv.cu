#include "hip/hip_runtime.h"
// GEMV: GEneric Matrix-Vector product
// Compute y = Ax, where
// A is a [M, N] matrix, x is a [N] vector, and y is a [M] vector

#include "common.h"

template <int block_size>
__global__ void hgemv_cuda_kernel(const half *__restrict__ A, const half *__restrict__ x, half *__restrict__ y, int M,
                                  int N) {
    float sum = 0.f;

    for (int j = threadIdx.x * 8; j < N; j += blockDim.x * 8) {
        float4 A_h8 = *(float4 *)&A[blockIdx.x * N + j];
        float4 x_h8 = *(float4 *)&x[j];

#pragma unroll
        for (int i = 0; i < 4; i++) {
            float2 f2 = __half22float2(__hmul2(((half2 *)&A_h8)[i], ((half2 *)&x_h8)[i]));
            sum += f2.x + f2.y;
        }
    }

    sum = block_reduce_sum<block_size, false>(sum);

    if (threadIdx.x == 0) {
        y[blockIdx.x] = sum;
    }
}

static inline void hgemv_cuda(const half *A, const half *x, half *y, int M, int N) {
    constexpr int num_threads = 512;
    const int num_blocks = M;
    hgemv_cuda_kernel<num_threads><<<num_blocks, num_threads>>>(A, x, y, M, N);
}

static inline void hgemv_cublas(hipblasHandle_t handle, const half *A, const half *x, half *y, int M, int N) {
    const half alpha = 1;
    const half beta = 0;
    CHECK_CUBLAS(hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, 1, N, &alpha, A, N, x, N, &beta, y, M));
}

int main() {
    constexpr size_t M = 4096;
    constexpr size_t N = 4096 * 4;

    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    half *h_A, *h_x, *h_y, *h_y_ref;
    CHECK_CUDA(hipHostMalloc(&h_A, M * N * sizeof(half), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_x, N * sizeof(half), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_y, M * sizeof(half), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_y_ref, M * sizeof(half), hipHostMallocDefault));

    half *d_A, *d_x, *d_y, *d_y_ref;
    CHECK_CUDA(hipMalloc(&d_A, M * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_x, N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_y, M * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_y_ref, M * sizeof(half)));

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i] = uniform(-0.5, 0.5);
        }
    }
    for (int i = 0; i < N; i++) {
        h_x[i] = uniform(-0.5, 0.5);
    }
    CHECK_CUDA(hipMemcpyAsync(d_A, h_A, M * N * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(d_x, h_x, N * sizeof(half), hipMemcpyHostToDevice));

    // cuda implementation
    hgemv_cuda(d_A, d_x, d_y, M, N);
    CHECK_CUDA(hipMemcpy(h_y, d_y, M * sizeof(half), hipMemcpyDeviceToHost));

    // cublas implementation
    hgemv_cublas(cublas_handle, d_A, d_x, d_y_ref, M, N);
    CHECK_CUDA(hipMemcpy(h_y_ref, d_y_ref, M * sizeof(half), hipMemcpyDeviceToHost));

    // check results
    check_is_close(h_y, h_y_ref, M, 1e-2, 1e-3);

    // benchmark
    constexpr float nbytes = (M * N + M + N) * sizeof(half);
    const float elapsed_cuda = timeit([=] { hgemv_cuda(d_A, d_x, d_y, M, N); }, 2, 10);
    const float elapsed_cublas = timeit([=] { hgemv_cublas(cublas_handle, d_A, d_x, d_y_ref, M, N); }, 2, 10);

    printf("[hgemv-cuda]   elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed_cuda * 1e6f, nbytes / 1e9f / elapsed_cuda);
    printf("[hgemv-cublas] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed_cublas * 1e6f,
           nbytes / 1e9f / elapsed_cublas);

    CHECK_CUBLAS(hipblasDestroy(cublas_handle));

    CHECK_CUDA(hipHostFree(h_A));
    CHECK_CUDA(hipHostFree(h_x));
    CHECK_CUDA(hipHostFree(h_y));
    CHECK_CUDA(hipHostFree(h_y_ref));

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUDA(hipFree(d_y_ref));

    return 0;
}
