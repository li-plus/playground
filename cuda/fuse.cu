#include "hip/hip_runtime.h"
#include "common.h"

// separate mul & add kernels
__global__ void mul_cuda_kernel(const float *input, const float *other, float *output) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    output[idx] = input[idx] * other[idx];
}

__global__ void add_cuda_kernel(const float *input, const float *other, float *output) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    output[idx] = input[idx] + other[idx];
}

void naive_mul_add_cuda(const float *input, const float *alpha, const float *beta, float *output, int n) {
    mul_cuda_kernel<<<n / 128, 128>>>(input, alpha, output);
    add_cuda_kernel<<<n / 128, 128>>>(output, beta, output);
}

// fused mul & add
__global__ void fused_mul_add_cuda_kernel(const float *input, const float *alpha, const float *beta, float *output) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    output[idx] = std::fma(input[idx], alpha[idx], beta[idx]);
}

void fused_mul_add_cuda(const float *input, const float *alpha, const float *beta, float *output, int n) {
    fused_mul_add_cuda_kernel<<<n / 128, 128>>>(input, alpha, beta, output);
}

int main() {
    const int n = 1024 * 256;

    float *h_input = (float *)malloc(n * sizeof(float));
    float *h_alpha = (float *)malloc(n * sizeof(float));
    float *h_beta = (float *)malloc(n * sizeof(float));
    float *h_output_naive = (float *)malloc(n * sizeof(float));
    float *h_output_fused = (float *)malloc(n * sizeof(float));

    float *d_input, *d_alpha, *d_beta, *d_output_naive, *d_output_fused;

    CHECK_CUDA(hipMalloc(&d_input, n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_alpha, n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_beta, n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_naive, n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_fused, n * sizeof(float)));

    for (int i = 0; i < n; i++) {
        h_input[i] = uniform();
        h_alpha[i] = uniform();
        h_beta[i] = uniform();
    }

    CHECK_CUDA(hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_alpha, h_alpha, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_beta, h_beta, n * sizeof(float), hipMemcpyHostToDevice));

    fused_mul_add_cuda(d_input, d_alpha, d_beta, d_output_fused, n);
    CHECK_CUDA(hipMemcpy(h_output_fused, d_output_fused, n * sizeof(float), hipMemcpyDeviceToHost));

    naive_mul_add_cuda(d_input, d_alpha, d_beta, d_output_naive, n);
    CHECK_CUDA(hipMemcpy(h_output_naive, d_output_naive, n * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipDeviceSynchronize());

    // check correctness
    for (int i = 0; i < n; i++) {
        CHECK(is_close(h_output_fused[i], h_output_naive[i]))
            << h_input[i] << " * " << h_alpha[i] << " + " << h_beta[i] << " = " << h_output_fused[i] << " vs "
            << h_output_naive[i];
    }

    const float naive_elapsed =
        timeit([=] { naive_mul_add_cuda(d_input, d_alpha, d_beta, d_output_naive, n); }, 100, 10000);
    const float fused_elapsed =
        timeit([=] { fused_mul_add_cuda(d_input, d_alpha, d_beta, d_output_fused, n); }, 100, 10000);

    printf("naive: %.3f us\n", naive_elapsed * 1e6f);
    printf("fused: %.3f us\n", fused_elapsed * 1e6f);

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_alpha));
    CHECK_CUDA(hipFree(d_beta));
    CHECK_CUDA(hipFree(d_output_naive));
    CHECK_CUDA(hipFree(d_output_fused));

    free(h_input);
    free(h_alpha);
    free(h_beta);
    free(h_output_naive);
    free(h_output_fused);

    return 0;
}