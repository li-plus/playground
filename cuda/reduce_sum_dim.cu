#include "hip/hip_runtime.h"
#include "common.h"

template <int block_size>
__global__ void sum_block_reduce_kernel(const float *__restrict__ input, float *__restrict__ output, int N) {
    const float *input_row = input + blockIdx.x * N;

    float sum = 0.f;
    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        sum += input_row[i];
    }
    sum = block_reduce_sum<block_size, false>(sum);

    if (threadIdx.x == 0) {
        output[blockIdx.x] = sum;
    }
}

hipError_t sum_block_reduce_cuda(const float *input, float *output, int M, int N) {
    constexpr int block_size = 128;
    const int grid_size = M;
    sum_block_reduce_kernel<block_size><<<grid_size, block_size>>>(input, output, N);
    return hipGetLastError();
}

template <int block_size>
__global__ void sum_warp_reduce_kernel(const float *__restrict__ input, float *__restrict__ output, int N) {
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    const int row_id = blockIdx.x * (block_size / WARP_SIZE) + warp_id;
    const float *input_row = input + row_id * N;

    float sum = 0.f;
    for (int i = lane_id; i < N; i += WARP_SIZE) {
        sum += input_row[i];
    }
    sum = warp_reduce_sum(sum);

    if (lane_id == 0) {
        output[row_id] = sum;
    }
}

hipError_t sum_warp_reduce_cuda(const float *input, float *output, int M, int N) {
    constexpr int block_size = 128;
    const int grid_size = M / (block_size / WARP_SIZE);
    sum_warp_reduce_kernel<block_size><<<grid_size, block_size>>>(input, output, N);
    return hipGetLastError();
}

template <int block_size>
__global__ void sum_cg_warp_reduce_kernel(const float *__restrict__ input, float *__restrict__ output, int N) {
    auto block = cg::this_thread_block();
    auto tile = cg::tiled_partition<WARP_SIZE>(block);

    const int row_id = block.group_index().x * tile.meta_group_size() + tile.meta_group_rank();
    const float *input_row = input + row_id * N;

    float sum = 0.f;
    // replacing WARP_SIZE with tile.num_threads() will disable nvcc loop unrolling, causing bad performance
    for (int i = tile.thread_rank(); i < N; i += WARP_SIZE) {
        sum += input_row[i];
    }
    sum = cg::reduce(tile, sum, cg::plus<float>());

    if (tile.thread_rank() == 0) {
        output[row_id] = sum;
    }
}

hipError_t sum_cg_warp_reduce_cuda(const float *input, float *output, int M, int N) {
    constexpr int block_size = 128;
    const int grid_size = M / (block_size / WARP_SIZE);
    sum_cg_warp_reduce_kernel<block_size><<<grid_size, block_size>>>(input, output, N);
    return hipGetLastError();
}

int main() {
    constexpr size_t M = 512;
    constexpr size_t N = 2048;

    float *h_input, *h_output_expect, *h_output_actual;
    CHECK_CUDA(hipHostMalloc(&h_input, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_output_expect, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_output_actual, M * N * sizeof(float), hipHostMallocDefault));

    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, M * N * sizeof(float)));

    for (int i = 0; i < M * N; i++) {
        h_input[i] = uniform(-1, 1);
    }
    CHECK_CUDA(hipMemcpyAsync(d_input, h_input, M * N * sizeof(float), hipMemcpyHostToDevice));

    // run & check
    {
        CHECK_CUDA(hipMemsetAsync(d_output, 0, M * N * sizeof(float)));
        CHECK_CUDA(sum_block_reduce_cuda(d_input, d_output, M, N));
        CHECK_CUDA(hipMemcpy(h_output_expect, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));
    }

    auto run_and_check = [&](decltype(sum_block_reduce_cuda) fn) {
        CHECK_CUDA(hipMemsetAsync(d_output, 0, M * N * sizeof(float)));
        CHECK_CUDA(fn(d_input, d_output, M, N));
        CHECK_CUDA(hipMemcpy(h_output_actual, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));
        check_is_close(h_output_expect, h_output_actual, M * N, 1e-3, 1e-3);
    };
    run_and_check(sum_warp_reduce_cuda);
    run_and_check(sum_cg_warp_reduce_cuda);

    // benchmark
    auto benchmark = [&](decltype(sum_block_reduce_cuda) fn, const char *name) {
        constexpr float nbytes = 2 * M * N * sizeof(float);
        const float elapsed = timeit([&] { CHECK_CUDA(fn(d_input, d_output, M, N)); }, 10, 100);
        printf("[%s] elapsed %.3f us, bandwidth %.3f GB/s\n", name, elapsed * 1e6f, nbytes / 1e9f / elapsed);
    };
    benchmark(sum_block_reduce_cuda, "block-reduce");
    benchmark(sum_warp_reduce_cuda, "warp-reduce");
    benchmark(sum_cg_warp_reduce_cuda, "cg-warp-reduce");

    CHECK_CUDA(hipHostFree(h_input));
    CHECK_CUDA(hipHostFree(h_output_expect));
    CHECK_CUDA(hipHostFree(h_output_actual));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));

    return 0;
}
