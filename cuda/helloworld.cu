#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>

__global__ void add(float *A, float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

static void launch_add(float *A, float *B, float *C, int N) {
    constexpr int num_threads = 1024;
    const int num_blocks = N / num_threads;
    add<<<num_blocks, num_threads>>>(A, B, C, N);
}

static void ref_add(float *A, float *B, float *C, int N) {
    for (int i = 0; i < N; i++) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    constexpr size_t MB = 1024ull * 1024ull;
    constexpr size_t GB = 1024ull * MB;

    constexpr size_t N = 1 * GB;

    float *hA, *hB, *hC;
    CHECK_CUDA(hipHostAlloc(&hA, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&hB, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&hC, N * sizeof(float), hipHostMallocDefault));

    for (int i = 0; i < N; i++) {
        hA[i] = i;
        hB[i] = i - 1;
    }

    float *dA, *dB, *dC;
    CHECK_CUDA(hipMalloc(&dA, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&dB, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&dC, N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(dA, hA, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, hB, N * sizeof(float), hipMemcpyHostToDevice));

    // launch cuda kernel
    launch_add(dA, dB, dC, N);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(hC, dC, N * sizeof(float), hipMemcpyDeviceToHost));

    // run cpu ref impl
    float *ref_C = (float *)malloc(N * sizeof(float));
    ref_add(hA, hB, ref_C, N);

    // check results
    for (int i = 0; i < N; i++) {
        if (!is_close(hC[i], ref_C[i])) {
            printf("value diff: %f vs %f\n", hC[i], ref_C[i]);
        }
    }

    free(ref_C);

    // perf
    auto fn = [=] { launch_add(dA, dB, dC, N); };
    float elapsed_ms = timeit(fn, 2, 10);
    float bw_peak = 900; // V100 900GB/s
    float bw_actual = 3 * N * sizeof(float) / (float) GB / (elapsed_ms / 1000);
    float bw_util = bw_actual / bw_peak;
    printf("elapsed %.3f ms, bandwidth %.3f GB/s / peak %.3f GB/s (%.3f%%)\n", elapsed_ms, bw_actual, bw_peak,
           bw_util * 100);

    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));

    CHECK_CUDA(hipHostFree(hA));
    CHECK_CUDA(hipHostFree(hB));
    CHECK_CUDA(hipHostFree(hC));
    return 0;
}
