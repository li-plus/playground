#include "hip/hip_runtime.h"
#include "common.h"
#include <hipDNN.h>

#define CHECK_CUDNN(expr)                                                                                              \
    do {                                                                                                               \
        hipdnnStatus_t status = (expr);                                                                                 \
        CHECK(status == HIPDNN_STATUS_SUCCESS) << "cudnn error: " << hipdnnGetErrorString(status);                       \
    } while (false)

template <int block_size>
__global__ void softmax_kernel(const float *__restrict__ input, float *__restrict__ output, int N) {
    const float *input_row = input + blockIdx.x * N;
    float *output_row = output + blockIdx.x * N;

    float max_val = -INFINITY;
    for (int i = threadIdx.x; i < N; i += block_size) {
        max_val = fmaxf(max_val, input_row[i]);
    }
    max_val = block_reduce_max<block_size, true>(max_val);

    float sum = 0.f;
    for (int i = threadIdx.x; i < N; i += block_size) {
        sum += expf(input_row[i] - max_val);
    }
    sum = block_reduce_sum<block_size, true>(sum);

    const float inv_sum = 1.f / sum;
    for (int i = threadIdx.x; i < N; i += block_size) {
        output_row[i] = expf(input_row[i] - max_val) * inv_sum;
    }
}

hipError_t softmax_forward_cuda(const float *input, float *output, int M, int N) {
    constexpr int block_size = 256;
    const int grid_size = M;
    softmax_kernel<block_size><<<grid_size, block_size>>>(input, output, N);
    return hipGetLastError();
}

template <int block_size>
__global__ void softmax_online_forward_kernel(const float *__restrict__ input, float *__restrict__ output, int N) {
    const float *input_row = input + blockIdx.x * N;
    float *output_row = output + blockIdx.x * N;

    float m = -INFINITY;
    float d = 0.f;
    for (int i = threadIdx.x; i < N; i += block_size) {
        const float x = input_row[i];
        const float m_old = m;
        m = fmaxf(m, x);
        d = expf(m_old - m) * d + expf(x - m);
    }
    const float m_local = m;
    m = block_reduce_max<block_size, true>(m_local);
    d = block_reduce_sum<block_size, true>(d * expf(m_local - m));

    const float inv_sum = 1.f / d;
    for (int i = threadIdx.x; i < N; i += block_size) {
        output_row[i] = expf(input_row[i] - m) * inv_sum;
    }
}

hipError_t softmax_online_forward_cuda(const float *input, float *output, int M, int N) {
    constexpr int block_size = 256;
    const int grid_size = M;
    softmax_online_forward_kernel<block_size><<<grid_size, block_size>>>(input, output, N);
    return hipGetLastError();
}

hipdnnStatus_t softmax_forward_cudnn(hipdnnHandle_t handle, const float *input, float *output, int M, int N) {
    hipdnnTensorDescriptor_t x_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&x_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));

    hipdnnTensorDescriptor_t y_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));

    const float alpha = 1.f;
    const float beta = 0.f;
    hipdnnStatus_t status = hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha,
                                               x_desc, input, &beta, y_desc, output);

    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(x_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(y_desc));

    return status;
}

template <int block_size>
__global__ void softmax_backward_kernel(const float *__restrict__ grad_output, const float *__restrict__ output,
                                        float *__restrict__ grad_input, int N) {
    const float *grad_output_row = grad_output + blockIdx.x * N;
    const float *output_row = output + blockIdx.x * N;
    float *grad_input_row = grad_input + blockIdx.x * N;

    float y_dot_dy = 0.f;
    for (int i = threadIdx.x; i < N; i += block_size) {
        y_dot_dy += grad_output_row[i] * output_row[i];
    }
    y_dot_dy = block_reduce_sum<block_size, true>(y_dot_dy);

    for (int i = threadIdx.x; i < N; i += block_size) {
        grad_input_row[i] = output_row[i] * (grad_output_row[i] - y_dot_dy);
    }
}

hipError_t softmax_backward_cuda(const float *grad_output, const float *output, float *grad_input, int M, int N) {
    constexpr int block_size = 256;
    const int grid_size = M;
    softmax_backward_kernel<block_size><<<grid_size, block_size>>>(grad_output, output, grad_input, N);
    return hipGetLastError();
}

hipdnnStatus_t softmax_backward_cudnn(hipdnnHandle_t handle, const float *grad_output, const float *output,
                                     float *grad_input, int M, int N) {
    hipdnnTensorDescriptor_t y_desc, dy_desc, dx_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&dy_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(dy_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&dx_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(dx_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));

    const float alpha = 1.f;
    const float beta = 0.f;
    hipdnnStatus_t status = hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha,
                                                y_desc, output, dy_desc, grad_output, &beta, dx_desc, grad_input);

    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(y_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(dy_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(dx_desc));

    return status;
}

void run_softmax_forward(hipdnnHandle_t handle, int M, int N) {
    float *h_x, *h_y_expect, *h_y_actual;
    CHECK_CUDA(hipHostMalloc(&h_x, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_y_expect, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_y_actual, M * N * sizeof(float), hipHostMallocDefault));

    float *d_x, *d_y;
    CHECK_CUDA(hipMalloc(&d_x, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_y, M * N * sizeof(float)));

    // initialize x
    for (int i = 0; i < M * N; i++) {
        h_x[i] = uniform();
    }
    CHECK_CUDA(hipMemcpy(d_x, h_x, M * N * sizeof(float), hipMemcpyHostToDevice));

    // cuda forward
    CHECK_CUDA(hipMemsetAsync(d_y, 0, M * N * sizeof(float)));
    CHECK_CUDA(softmax_forward_cuda(d_x, d_y, M, N));
    CHECK_CUDA(hipMemcpy(h_y_expect, d_y, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // online forward
    CHECK_CUDA(hipMemsetAsync(d_y, 0, M * N * sizeof(float)));
    CHECK_CUDA(softmax_online_forward_cuda(d_x, d_y, M, N));
    CHECK_CUDA(hipMemcpy(h_y_actual, d_y, M * N * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_y_expect, h_y_actual, M * N);

    // cudnn forward
    CHECK_CUDA(hipMemsetAsync(d_y, 0, M * N * sizeof(float)));
    CHECK_CUDNN(softmax_forward_cudnn(handle, d_x, d_y, M, N));
    CHECK_CUDA(hipMemcpy(h_y_actual, d_y, M * N * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_y_expect, h_y_actual, M * N);

    // benchmark forward
    printf("===== forward =====\n");
    {
        const float elapsed = timeit([&] { CHECK_CUDA(softmax_forward_cuda(d_x, d_y, M, N)); }, 10, 100);
        printf("[cuda] elapsed %.3f us\n", elapsed * 1e6);
    }
    {
        const float elapsed = timeit([&] { CHECK_CUDA(softmax_online_forward_cuda(d_x, d_y, M, N)); }, 10, 100);
        printf("[online] elapsed %.3f us\n", elapsed * 1e6);
    }
    {
        const float elapsed = timeit([&] { CHECK_CUDNN(softmax_forward_cudnn(handle, d_x, d_y, M, N)); }, 10, 100);
        printf("[cudnn] elapsed %.3f us\n", elapsed * 1e6);
    }
    // [cuda] elapsed 10.383 us
    // [online] elapsed 11.233 us
    // [cudnn] elapsed 13.445 us

    // clean up
    CHECK_CUDA(hipHostFree(h_x));
    CHECK_CUDA(hipHostFree(h_y_expect));
    CHECK_CUDA(hipHostFree(h_y_actual));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
}

void run_softmax_backward(hipdnnHandle_t handle, int M, int N) {
    float *h_y, *h_dy, *h_dx_expect, *h_dx_actual;
    CHECK_CUDA(hipHostMalloc(&h_y, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_dy, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_dx_expect, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_dx_actual, M * N * sizeof(float), hipHostMallocDefault));

    float *d_y, *d_dy, *d_dx;
    CHECK_CUDA(hipMalloc(&d_y, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dy, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dx, M * N * sizeof(float)));

    // initialize
    for (int i = 0; i < M * N; i++) {
        h_y[i] = uniform(-1.0, 1.0);
        h_dy[i] = uniform(-1.0, 1.0);
    }
    CHECK_CUDA(hipMemcpyAsync(d_y, h_y, M * N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(d_dy, h_dy, M * N * sizeof(float), hipMemcpyHostToDevice));

    // cuda backward
    CHECK_CUDA(hipMemsetAsync(d_dx, 0, M * N * sizeof(float)));
    CHECK_CUDA(softmax_backward_cuda(d_dy, d_y, d_dx, M, N));
    CHECK_CUDA(hipMemcpy(h_dx_expect, d_dx, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // cudnn backward
    CHECK_CUDA(hipMemsetAsync(d_dx, 0, M * N * sizeof(float)));
    CHECK_CUDNN(softmax_backward_cudnn(handle, d_dy, d_y, d_dx, M, N));
    CHECK_CUDA(hipMemcpy(h_dx_actual, d_dx, M * N * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_dx_expect, h_dx_actual, M * N);

    // benchmark backward
    printf("===== backward =====\n");
    {
        const float elapsed = timeit([&] { CHECK_CUDA(softmax_backward_cuda(d_dy, d_y, d_dx, M, N)); }, 10, 100);
        printf("[cuda] elapsed %.3f us\n", elapsed * 1e6);
    }
    {
        const float elapsed =
            timeit([&] { CHECK_CUDNN(softmax_backward_cudnn(handle, d_dy, d_y, d_dx, M, N)); }, 10, 100);
        printf("[cudnn] elapsed %.3f us\n", elapsed * 1e6);
    }
    // [cuda] elapsed 10.168 us
    // [cudnn] elapsed 12.595 us

    // clean up
    CHECK_CUDA(hipHostFree(h_y));
    CHECK_CUDA(hipHostFree(h_dy));
    CHECK_CUDA(hipHostFree(h_dx_expect));
    CHECK_CUDA(hipHostFree(h_dx_actual));
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUDA(hipFree(d_dy));
    CHECK_CUDA(hipFree(d_dx));
}

int main() {
    constexpr int M = 1024;
    constexpr int N = 2048;

    hipdnnHandle_t handle;
    CHECK_CUDNN(hipdnnCreate(&handle));

    run_softmax_forward(handle, M, N);
    run_softmax_backward(handle, M, N);

    CHECK_CUDNN(hipdnnDestroy(handle));

    return 0;
}