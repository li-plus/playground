#include "hip/hip_runtime.h"
#include "common.h"
#include <hipDNN.h>

#define CHECK_CUDNN(call)                                                                                              \
    do {                                                                                                               \
        hipdnnStatus_t status = (call);                                                                                 \
        CHECK(status == HIPDNN_STATUS_SUCCESS) << "cudnn error: " << hipdnnGetErrorString(status);                       \
    } while (false)

template <int block_size>
__global__ void softmax_kernel(const float *input, float *output, int N) {
    const float *input_row = input + blockIdx.x * N;
    float *output_row = output + blockIdx.x * N;

    float max_val = -INFINITY;
    for (int i = threadIdx.x; i < N; i += block_size) {
        max_val = fmaxf(max_val, input_row[i]);
    }
    max_val = block_reduce_max<block_size, true>(max_val);

    float sum = 0.f;
    for (int i = threadIdx.x; i < N; i += block_size) {
        sum += expf(input_row[i] - max_val);
    }
    sum = block_reduce_sum<block_size, true>(sum);

    const float inv_sum = 1.f / sum;
    for (int i = threadIdx.x; i < N; i += block_size) {
        output_row[i] = expf(input_row[i] - max_val) * inv_sum;
    }
}

void softmax_cuda(const float *input, float *output, int M, int N) {
    constexpr int block_size = 256;
    const int grid_size = M;
    softmax_kernel<block_size><<<grid_size, block_size>>>(input, output, N);
    CHECK_CUDA(hipGetLastError());
}

int main() {
    constexpr int M = 1024;
    constexpr int N = 2048;

    hipdnnHandle_t handle;
    CHECK_CUDNN(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t x_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&x_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));

    hipdnnTensorDescriptor_t y_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, M, N, 1, 1));

    float *h_x, *h_y_cuda, *h_y_cudnn;
    CHECK_CUDA(hipHostMalloc(&h_x, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_y_cuda, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_y_cudnn, M * N * sizeof(float), hipHostMallocDefault));

    float *d_x, *d_y;
    CHECK_CUDA(hipMalloc(&d_x, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_y, M * N * sizeof(float)));

    // initialize x
    for (int i = 0; i < M * N; i++) {
        h_x[i] = uniform();
    }
    CHECK_CUDA(hipMemcpy(d_x, h_x, M * N * sizeof(float), hipMemcpyHostToDevice));

    // cuda
    CHECK_CUDA(hipMemsetAsync(d_y, 0, M * N * sizeof(float)));
    softmax_cuda(d_x, d_y, M, N);
    CHECK_CUDA(hipMemcpy(h_y_cuda, d_y, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // cudnn
    const float alpha = 1.f;
    const float beta = 0.f;
    CHECK_CUDA(hipMemsetAsync(d_y, 0, M * N * sizeof(float)));
    hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, x_desc, d_x, &beta, y_desc,
                        d_y);
    CHECK_CUDA(hipMemcpy(h_y_cudnn, d_y, M * N * sizeof(float), hipMemcpyDeviceToHost));

    check_is_close(h_y_cuda, h_y_cudnn, M * N, 1e-4f);

    // benchmark
    {
        const float elapsed = timeit([&] { softmax_cuda(d_x, d_y, M, N); }, 10, 100);
        printf("[cuda] elapsed %.3f us\n", elapsed * 1e6);
    }
    {
        const float elapsed = timeit(
            [&] {
                hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, x_desc, d_x,
                                    &beta, y_desc, d_y);
            },
            10, 100);
        printf("[cudnn] elapsed %.3f us\n", elapsed * 1e6);
    }

    // clean up
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(x_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(y_desc));

    CHECK_CUDA(hipHostFree(h_x));
    CHECK_CUDA(hipHostFree(h_y_cuda));
    CHECK_CUDA(hipHostFree(h_y_cudnn));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));

    CHECK_CUDNN(hipdnnDestroy(handle));

    return 0;
}