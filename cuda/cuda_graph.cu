#include "hip/hip_runtime.h"
#include "common.h"

__global__ void short_kernel(const float *d_in, float *d_out, int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        d_out[idx] = 1.23 * d_in[idx];
    }
}

hipError_t short_kernel_launch(const float *d_in, float *d_out, int N, hipStream_t stream) {
    const int block_size = 128;
    const int grid_size = N / block_size;
    short_kernel<<<grid_size, block_size, 0, stream>>>(d_in, d_out, N);
    return hipGetLastError();
}

void work(const float *d_in, float *d_out, int N, int NUM_KERNELS, hipStream_t stream) {
    for (int i = 0; i < NUM_KERNELS; i++) {
        CHECK_CUDA(short_kernel_launch(d_in, d_out, N, stream));
    }
}

int main() {
    const int N = 1024;
    float *d_in, *d_ref, *d_out;
    CHECK_CUDA(hipMalloc(&d_in, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_ref, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_out, N * sizeof(float)));

    CHECK_CUDA(hipMemset(d_in, 0, N * sizeof(float)));

    const int NUM_KERNELS = 2048;

    const float naive_cost = timeit([=] { work(d_in, d_ref, N, NUM_KERNELS, hipStreamDefault); }, 2, 10);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    hipGraph_t graph = nullptr;
    hipGraphExec_t instance = nullptr;
    CHECK_CUDA(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
    work(d_in, d_out, N, NUM_KERNELS, stream);
    CHECK_CUDA(hipStreamEndCapture(stream, &graph));
    CHECK_CUDA(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));

    const float graph_cost = timeit([&] { CHECK_CUDA(hipGraphLaunch(instance, hipStreamDefault)); }, 2, 10);

    check_is_close_d(d_ref, d_out, N);

    printf("naive_cost: %.3f ms, graph_cost: %.3f ms, speedup: %.2f x\n", naive_cost * 1e3, graph_cost * 1e3,
           naive_cost / graph_cost);

    CHECK_CUDA(hipFree(d_in));
    CHECK_CUDA(hipFree(d_ref));
    CHECK_CUDA(hipFree(d_out));

    return 0;
}
