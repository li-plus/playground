#include "hip/hip_runtime.h"
#include "common.h"

__device__ __forceinline__ float warp_reduce_sum(float v) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        v += __shfl_xor_sync(0xffffffff, v, mask, warpSize);
    }
    return v;
}

__device__ __forceinline__ float block_reduce_sum(float v) {
    v = warp_reduce_sum(v);
    if (blockDim.x > warpSize) {
        __shared__ float shm[32];
        const int num_warps = blockDim.x / warpSize;
        const int warp_id = threadIdx.x / warpSize;
        const int lane_id = threadIdx.x % warpSize;
        if (lane_id == 0) {
            shm[warp_id] = v;
        }
        __syncthreads();
        v = warp_reduce_sum((lane_id < num_warps) ? shm[lane_id] : 0.f);
    }
    return v;
}

__global__ void sum_cuda_kernel(const float *input, float *output, float *reduce_buffer, int *semaphore, int N) {
    float sum = 0.f;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridDim.x * blockDim.x) {
        sum += input[i];
    }
    sum = block_reduce_sum(sum);

    __shared__ bool is_last_block_done_shared;

    if (threadIdx.x == 0) {
        reduce_buffer[blockIdx.x] = sum;
        __threadfence();
        const int prev_blocks_finished = atomicAdd(semaphore, 1);
        is_last_block_done_shared = (prev_blocks_finished == gridDim.x - 1);
    }
    __syncthreads();

    if (is_last_block_done_shared) {
        sum = 0.f;
        for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) {
            sum += reduce_buffer[i];
        }
        sum = block_reduce_sum(sum);

        if (threadIdx.x == 0) {
            *output = sum;
        }
    }
}

static void sum_cuda(const float *input, float *output, float *reduce_buffer, int *semaphore, int N, int num_blocks,
                     int num_threads) {
    CHECK_CUDA(hipMemsetAsync(semaphore, 0, sizeof(int)));
    sum_cuda_kernel<<<num_blocks, num_threads>>>(input, output, reduce_buffer, semaphore, N);
}

static float sum_cpu(const float *input, int N) {
    double sum = 0.f;
    for (int i = 0; i < N; i++) {
        sum += input[i];
    }
    return sum;
}

int main() {
    constexpr size_t N = 128ull * 1024 * 1024;
    constexpr int num_threads = 1024;
    constexpr int num_blocks = (N + num_threads - 1) / num_threads / 64;

    float *h_input;
    CHECK_CUDA(hipHostAlloc(&h_input, N * sizeof(float), hipHostMallocDefault));

    float *d_input, *d_output, *d_reduce_buffer;
    int *d_semaphore;
    CHECK_CUDA(hipMalloc(&d_input, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_reduce_buffer, num_blocks * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_semaphore, sizeof(int)));

    float h_output_cpu, h_output;

    for (int i = 0; i < N; i++) {
        h_input[i] = uniform(-1, 1);
    }
    h_output_cpu = sum_cpu(h_input, N);

    CHECK_CUDA(hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice));

    sum_cuda(d_input, d_output, d_reduce_buffer, d_semaphore, N, num_blocks, num_threads);
    CHECK_CUDA(hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost));

    // check diff
    CHECK(is_close(h_output, h_output_cpu, 1e-3, 1e-3)) << h_output << " vs " << h_output_cpu;

    // benchmark
    const float elapsed =
        timeit([=] { sum_cuda(d_input, d_output, d_reduce_buffer, d_semaphore, N, num_blocks, num_threads); }, 2, 10);
    const float bandwidth = N * sizeof(float) / 1e9 / elapsed;
    printf("[reduce_sum] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed * 1e6, bandwidth);

    CHECK_CUDA(hipHostFree(h_input));
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDA(hipFree(d_reduce_buffer));
    CHECK_CUDA(hipFree(d_semaphore));

    return 0;
}
