#include "common.h"
#include <hip/hip_runtime.h>
#include <string>
#include <vector>

int main() {
    constexpr size_t N = 256 * MB;

    float *hA, *hB;
    CHECK_CUDA(hipHostAlloc(&hA, N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&hB, N * sizeof(float), hipHostMallocDefault));

    float *dA, *dB;
    CHECK_CUDA(hipMalloc(&dA, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&dB, N * sizeof(float)));

    auto bench_fn = [=] {
        hipMemcpyAsync(dA, hA, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyAsync(hB, dB, N * sizeof(float), hipMemcpyDeviceToHost);
    };
    float elapsed_ms = timeit(bench_fn, 2, 10);
    constexpr float PCIE_BW = 64; // uni-directional 64GB/s
    float bw_actual = 2 * N * sizeof(float) / 1e9 / (elapsed_ms / 1e3);
    float bw_util = bw_actual / PCIE_BW;
    printf("pcie bandwidth %.2f / %.2f GB/s (%.2f%%)\n", bw_actual, PCIE_BW, bw_util * 100);

    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipHostFree(hA));
    CHECK_CUDA(hipHostFree(hB));

    return 0;
}
