#include "hip/hip_runtime.h"
#include "common.h"

__global__ void loop_cuda_kernel() {
    while (true) {
    }
}

void loop_cuda() { loop_cuda_kernel<<<1, 1>>>(); }

int main() {
    loop_cuda();
    printf("infinite loop\n");
    CHECK_CUDA(hipDeviceSynchronize());
    return 0;
}
