#include "hip/hip_runtime.h"
#include "common.h"

__global__ void im2col_kernel(const float *__restrict__ im, float *__restrict__ col, int N, int C, int IH, int IW,
                              int KH, int KW, int OH, int OW, int PH, int PW) {
    const int numel = N * OH * OW * KH * KW * C;
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < numel; idx += blockDim.x * gridDim.x) {
        const int c = idx % C;
        const int kw = (idx / C) % KW;
        const int kh = (idx / (C * KW)) % KH;
        const int ow = (idx / (C * KW * KH)) % OW;
        const int oh = (idx / (C * KW * KH * OW)) % OH;
        const int n = (idx / (C * KW * KH * OW * OH)) % N;

        // comply with cudnn convention: top left pixel is multiplied by bottom right filter element
        const int ih = oh - PH + (KH - 1 - kh);
        const int iw = ow - PW + (KW - 1 - kw);
        if (0 <= ih && ih < IH && 0 <= iw && iw < IW) {
            const int im_idx = ((n * IH + ih) * IW + iw) * C + c;
            col[idx] = im[im_idx];
        } else {
            col[idx] = 0.f;
        }
    }
}

hipError_t im2col_cuda(const float *im, float *col, int N, int IC, int IH, int IW, int KH, int KW, int OH, int OW,
                        int PH, int PW) {
    // im: [N, IH, IW, IC]
    // col: [N, OH, OW, KH, KW, IC]
    const int block_size = 128;
    const int grid_size = (N * OH * OW * KH * KW * IC + block_size - 1) / block_size;
    im2col_kernel<<<grid_size, block_size>>>(im, col, N, IC, IH, IW, KH, KW, OH, OW, PH, PW);
    return hipGetLastError();
}

hipblasStatus_t conv_cublas(hipblasHandle_t handle, const float *x, const float *w, float *y, void *workspace,
                           size_t workspace_size, int N, int IC, int OC, int IH, int IW, int KH, int KW, int OH, int OW,
                           int PH, int PW) {
    const size_t workspace_size_requested = N * OH * OW * KH * KW * IC * sizeof(float);
    CHECK(workspace_size_requested <= workspace_size) << workspace_size_requested << " vs " << workspace_size;
    float *col = (float *)workspace;
    CHECK_CUDA(im2col_cuda(x, col, N, IC, IH, IW, KH, KW, OH, OW, PH, PW));

    const float alpha = 1.f;
    const float beta = 0.f;
    hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, OC, N * OH * OW, KH * KW * IC, &alpha, w,
                                        KH * KW * IC, col, KH * KW * IC, &beta, y, OC);

#if 0
    float *h_col, *h_im, *h_w, *h_y;
    CHECK_CUDA(hipHostMalloc(&h_col, workspace_size_requested));
    CHECK_CUDA(hipMemcpy(h_col, col, workspace_size_requested, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipHostMalloc(&h_im, N * IH * IW * IC * sizeof(float)));
    CHECK_CUDA(hipMemcpy(h_im, x, N * IH * IW * IC * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipHostMalloc(&h_w, OC * KH * KW * IC * sizeof(float)));
    CHECK_CUDA(hipMemcpy(h_w, w, OC * KH * KW * IC * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipHostMalloc(&h_y, N * OH * OW * OC * sizeof(float)));
    CHECK_CUDA(hipMemcpy(h_y, y, N * OH * OW * OC * sizeof(float), hipMemcpyDeviceToHost));

    printf("im:\n");
    for (int i = 0; i < N * IH; i++) {
        for (int iw = 0; iw < IW; iw++) {
            for (int c = 0; c < IC; c++) {
                printf("%s%.3f%s", (c == 0) ? "[" : ", ", h_im[(i * IW + iw) * IC + c], (c == IC - 1) ? "] " : "");
            }
        }
        printf("\n");
    }

    printf("col:\n");
    for (int i = 0; i < N * OH * OW; i++) {
        for (int j = 0; j < KH * KW * IC; j++) {
            printf("%.3f, ", h_col[i * KH * KW * IC + j]);
        }
        printf("\n");
    }

    printf("filter:\n");
    for (int i = 0; i < OC; i++) {
        for (int j = 0; j < KH * KW * IC; j++) {
            printf("%.3f, ", h_w[i * KH * KW * IC + j]);
        }
        printf("\n");
    }

    printf("cublas-output:\n");
    for (int oh = 0; oh < N * OH; oh++) {
        for (int ow = 0; ow < OW; ow++) {
            for (int c = 0; c < OC; c++) {
                printf("%s%.3f%s", (c == 0) ? "[" : ", ", h_y[(oh * OW + ow) * OC + c], (c == OC - 1) ? "] " : "");
            }
        }
        printf("\n");
    }

    CHECK_CUDA(hipHostFree(h_col));
    CHECK_CUDA(hipHostFree(h_im));
    CHECK_CUDA(hipHostFree(h_w));
    CHECK_CUDA(hipHostFree(h_y));
#endif

    return status;
}

hipdnnStatus_t conv_cudnn(hipdnnHandle_t handle, const float *x, const float *w, float *y, void *workspace,
                         size_t workspace_size, int N, int IC, int OC, int IH, int IW, int KH, int KW, int OH, int OW,
                         int PH, int PW) {
    hipdnnTensorDescriptor_t x_desc, y_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&x_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, N, IC, IH, IW));

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&y_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, N, OC, OH, OW));

    hipdnnFilterDescriptor_t w_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&w_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NHWC, OC, IC, KH, KW));

    const int SH = 1; // stride h
    const int SW = 1; // stride w
    const int DH = 1; // dilation h
    const int DW = 1; // dilation w
    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(
        hipdnnSetConvolution2dDescriptor(conv_desc, PH, PW, SH, SW, DH, DW, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    int out_n, out_c, out_h, out_w;
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(conv_desc, x_desc, w_desc, &out_n, &out_c, &out_h, &out_w));
    CHECK(out_n == N && out_c == OC && out_h == OH && out_w == OW);

    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    size_t workspace_size_requested;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, x_desc, w_desc, conv_desc, y_desc, algo,
                                                        &workspace_size_requested));
    CHECK(workspace_size_requested <= workspace_size) << workspace_size_requested << " vs " << workspace_size;

    const float alpha = 1.f;
    const float beta = 0.f;
    hipdnnStatus_t status = hipdnnConvolutionForward(handle, &alpha, x_desc, x, w_desc, w, conv_desc, algo, workspace,
                                                   workspace_size, &beta, y_desc, y);

    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(w_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(x_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(y_desc));

#if 0
    float *h_y;
    CHECK_CUDA(hipHostMalloc(&h_y, N * OH * OW * OC * sizeof(float)));
    CHECK_CUDA(hipMemcpy(h_y, y, N * OH * OW * OC * sizeof(float), hipMemcpyDeviceToHost));

    printf("cudnn-output:\n");
    for (int oh = 0; oh < N * OH; oh++) {
        for (int ow = 0; ow < OW; ow++) {
            for (int c = 0; c < OC; c++) {
                printf("%s%.3f%s", (c == 0) ? "[" : ", ", h_y[(oh * OW + ow) * OC + c], (c == OC - 1) ? "] " : "");
            }
        }
        printf("\n");
    }

    CHECK_CUDA(hipHostFree(h_y));
#endif

    return status;
}

int main() {
    hipdnnHandle_t cudnn_handle;
    CHECK_CUDNN(hipdnnCreate(&cudnn_handle));

    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    const int N = 2;
    const int IC = 4;
    const int OC = 8;
    const int IH = 128;
    const int IW = 256;
    const int KH = 3; // kernel h
    const int KW = 3; // kernel w
    const int PH = 1; // pad h
    const int PW = 1; // pad w
    const int OH = (IH + 2 * PH - KH + 1);
    const int OW = (IW + 2 * PW - KW + 1);

    printf("input  [%d, %d, %d, %d]\n", N, IH, IW, IC);
    printf("filter [%d, %d, %d, %d]\n", OC, KH, KW, IC);
    printf("output [%d, %d, %d, %d]\n", N, OH, OW, OC);

    float *h_x, *h_w, *h_y_actual, *h_y_expect;
    CHECK_CUDA(hipHostMalloc(&h_x, N * IH * IW * IC * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_w, OC * KH * KW * IC * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_y_actual, N * OH * OW * OC * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_y_expect, N * OH * OW * OC * sizeof(float)));

    float *d_x, *d_w, *d_y;
    CHECK_CUDA(hipMalloc(&d_x, N * IH * IW * IC * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_w, OC * KH * KW * IC * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_y, N * OH * OW * OC * sizeof(float)));

    for (int i = 0; i < N * IH * IW * IC; i++) {
        h_x[i] = uniform(-0.5, 0.5);
    }
    for (int i = 0; i < OC * KH * KW * IC; i++) {
        h_w[i] = uniform(-0.5, 0.5);
    }
    CHECK_CUDA(hipMemcpyAsync(d_x, h_x, N * IH * IW * IC * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(d_w, h_w, OC * KH * KW * IC * sizeof(float), hipMemcpyHostToDevice));

    void *workspace;
    size_t workspace_size = 16 * 1024 * 1024;
    CHECK_CUDA(hipMalloc(&workspace, workspace_size));

    CHECK_CUDA(hipMemsetAsync(d_y, 0, N * OH * OW * OC * sizeof(float)));
    CHECK_CUBLAS(conv_cublas(cublas_handle, d_x, d_w, d_y, workspace, workspace_size, N, IC, OC, IH, IW, KH, KW, OH, OW,
                             PH, PW));
    CHECK_CUDA(hipMemcpy(h_y_actual, d_y, N * OH * OW * OC * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipMemsetAsync(d_y, 0, N * OH * OW * OC * sizeof(float)));
    CHECK_CUDNN(
        conv_cudnn(cudnn_handle, d_x, d_w, d_y, workspace, workspace_size, N, IC, OC, IH, IW, KH, KW, OH, OW, PH, PW));
    CHECK_CUDA(hipMemcpy(h_y_expect, d_y, N * OH * OW * OC * sizeof(float), hipMemcpyDeviceToHost));

    check_is_close(h_y_expect, h_y_actual, N * OH * OW * OC, 1e-3f);

    {
        const float elapsed = timeit(
            [&] {
                CHECK_CUBLAS(conv_cublas(cublas_handle, d_x, d_w, d_y, workspace, workspace_size, N, IC, OC, IH, IW, KH,
                                         KW, OH, OW, PH, PW));
            },
            2, 10);
        printf("[im2col+cublas] elapsed %.3f us\n", elapsed * 1e6f);
    }
    {
        const float elapsed = timeit(
            [&] {
                CHECK_CUDNN(conv_cudnn(cudnn_handle, d_x, d_w, d_y, workspace, workspace_size, N, IC, OC, IH, IW, KH,
                                       KW, OH, OW, PH, PW));
            },
            2, 10);
        printf("[cudnn-conv] elapsed %.3f us\n", elapsed * 1e6f);
    }

    CHECK_CUDA(hipFree(workspace));
    CHECK_CUDA(hipHostFree(h_x));
    CHECK_CUDA(hipHostFree(h_w));
    CHECK_CUDA(hipHostFree(h_y_expect));
    CHECK_CUDA(hipHostFree(h_y_actual));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_w));
    CHECK_CUDA(hipFree(d_y));

    CHECK_CUBLAS(hipblasDestroy(cublas_handle));
    CHECK_CUDNN(hipdnnDestroy(cudnn_handle));

    return 0;
}