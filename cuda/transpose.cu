#include "hip/hip_runtime.h"
// https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/

#include "common.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>
#include <cuda/pipeline>

constexpr int TILE_DIM = 32;
constexpr int BLOCK_ROWS = 8;

__global__ void transpose_naive_kernel(float *odata, const float *idata, int M, int N) {
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[x * M + (y + j)] = idata[(y + j) * N + x];
}

__global__ void transpose_coalesced_kernel(float *odata, const float *idata, int M, int N) {
    __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // prevent bank conflict

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * N + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x; // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * M + x] = tile[threadIdx.x][threadIdx.y + j];
}

__global__ void transpose_swizzle_kernel(float *odata, const float *idata, int M, int N) {
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][(threadIdx.y + j) ^ threadIdx.x] = idata[(y + j) * N + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x; // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * M + x] = tile[threadIdx.x][threadIdx.x ^ (threadIdx.y + j)];
}

__global__ void transpose_async_barrier_swizzle_kernel(float *odata, const float *idata, int M, int N) {
    namespace cg = cooperative_groups;

    __shared__ float tile[TILE_DIM][TILE_DIM];

    __shared__ cuda::barrier<cuda::thread_scope_block> barrier;
    auto block = cg::this_thread_block();
    if (block.thread_rank() == 0) {
        init(&barrier, block.size()); // Initialize the barrier with expected arrival count
    }
    block.sync();

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        cuda::memcpy_async(&tile[threadIdx.y + j][(threadIdx.y + j) ^ threadIdx.x], &idata[(y + j) * N + x],
                           sizeof(float), barrier);

    x = blockIdx.y * TILE_DIM + threadIdx.x; // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    barrier.arrive_and_wait();

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * M + x] = tile[threadIdx.x][threadIdx.x ^ (threadIdx.y + j)];
}

__global__ void transpose_async_pipeline_swizzle_kernel(float *odata, const float *idata, int M, int N) {
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        cuda::memcpy_async(&tile[threadIdx.y + j][(threadIdx.y + j) ^ threadIdx.x], &idata[(y + j) * N + x],
                           sizeof(float), pipe);

    pipe.producer_commit();

    x = blockIdx.y * TILE_DIM + threadIdx.x; // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    pipe.consumer_wait();
    __syncthreads();

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * M + x] = tile[threadIdx.x][threadIdx.x ^ (threadIdx.y + j)];
}

#define make_launcher(launcher, kernel)                                                                                \
    void launcher(float *odata, const float *idata, int M, int N) {                                                    \
        const dim3 threads(TILE_DIM, BLOCK_ROWS);                                                                      \
        const dim3 blocks(N / TILE_DIM, M / TILE_DIM);                                                                 \
        kernel<<<blocks, threads>>>(odata, idata, M, N);                                                               \
    }

make_launcher(transpose_naive_cuda, transpose_naive_kernel);
make_launcher(transpose_coalesced_cuda, transpose_coalesced_kernel);
make_launcher(transpose_swizzle_cuda, transpose_swizzle_kernel);
make_launcher(transpose_async_barrier_swizzle_cuda, transpose_async_barrier_swizzle_kernel);
make_launcher(transpose_async_pipeline_swizzle_cuda, transpose_async_pipeline_swizzle_kernel);

#undef make_launcher

int main() {
    constexpr size_t M = 4096;
    constexpr size_t N = 4096;

    float *h_input, *h_output_ref, *h_output_out;
    CHECK_CUDA(hipHostAlloc(&h_input, M * N * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&h_output_ref, N * M * sizeof(float), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&h_output_out, N * M * sizeof(float), hipHostMallocDefault));

    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, N * M * sizeof(float)));

    // set inputs
    for (size_t i = 0; i < M * N; i++) {
        h_input[i] = uniform();
    }
    CHECK_CUDA(hipMemcpy(d_input, h_input, M * N * sizeof(float), hipMemcpyHostToDevice));

    // compute
    CHECK_CUDA(hipMemset(d_output, 0, N * M * sizeof(float)));
    transpose_naive_cuda(d_output, d_input, M, N);
    CHECK_CUDA(hipMemcpy(h_output_ref, d_output, N * M * sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipMemset(d_output, 0, N * M * sizeof(float)));
    transpose_coalesced_cuda(d_output, d_input, M, N);
    CHECK_CUDA(hipMemcpy(h_output_out, d_output, N * M * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_output_out, h_output_ref, N * M);

    CHECK_CUDA(hipMemset(d_output, 0, N * M * sizeof(float)));
    transpose_swizzle_cuda(d_output, d_input, M, N);
    CHECK_CUDA(hipMemcpy(h_output_out, d_output, N * M * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_output_out, h_output_ref, N * M);

    CHECK_CUDA(hipMemset(d_output, 0, N * M * sizeof(float)));
    transpose_async_barrier_swizzle_cuda(d_output, d_input, M, N);
    CHECK_CUDA(hipMemcpy(h_output_out, d_output, N * M * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_output_out, h_output_ref, N * M);

    CHECK_CUDA(hipMemset(d_output, 0, N * M * sizeof(float)));
    transpose_async_pipeline_swizzle_cuda(d_output, d_input, M, N);
    CHECK_CUDA(hipMemcpy(h_output_out, d_output, N * M * sizeof(float), hipMemcpyDeviceToHost));
    check_is_close(h_output_out, h_output_ref, N * M);

    // benchmark
    {
        const float elapsed = timeit([=] { transpose_naive_cuda(d_output, d_input, M, N); }, 2, 10);
        const float bandwidth = 2 * M * N * sizeof(float) / 1e9 / elapsed;
        printf("[naive] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed * 1e6, bandwidth);
    }
    {
        const float elapsed = timeit([=] { transpose_coalesced_cuda(d_output, d_input, M, N); }, 2, 10);
        const float bandwidth = 2 * M * N * sizeof(float) / 1e9 / elapsed;
        printf("[coalesced] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed * 1e6, bandwidth);
    }
    {
        const float elapsed = timeit([=] { transpose_swizzle_cuda(d_output, d_input, M, N); }, 2, 10);
        const float bandwidth = 2 * M * N * sizeof(float) / 1e9 / elapsed;
        printf("[swizzle] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed * 1e6, bandwidth);
    }
    {
        const float elapsed = timeit([=] { transpose_async_barrier_swizzle_cuda(d_output, d_input, M, N); }, 2, 10);
        const float bandwidth = 2 * M * N * sizeof(float) / 1e9 / elapsed;
        printf("[async-barrier] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed * 1e6, bandwidth);
    }
    {
        const float elapsed = timeit([=] { transpose_async_pipeline_swizzle_cuda(d_output, d_input, M, N); }, 2, 10);
        const float bandwidth = 2 * M * N * sizeof(float) / 1e9 / elapsed;
        printf("[async-pipeline] elapsed %.3f us, bandwidth %.3f GB/s\n", elapsed * 1e6, bandwidth);
    }

    CHECK_CUDA(hipHostFree(h_input));
    CHECK_CUDA(hipHostFree(h_output_ref));
    CHECK_CUDA(hipHostFree(h_output_out));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));

    return 0;
}
