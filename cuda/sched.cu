#include "hip/hip_runtime.h"
#include "common.h"

constexpr int N = 4096;
constexpr int BLOCK_SIZE = 128;

__global__ void interleaved_cuda_kernel(const float *__restrict__ input, float *__restrict__ output) {
#pragma unroll
    for (int s = 0; s < N; s += BLOCK_SIZE) {
        output[s + threadIdx.x] = logf(expf(cosf(sinf(input[s + threadIdx.x]))));
    }
}

void interleaved_cuda(const float *input, float *output) { interleaved_cuda_kernel<<<1, BLOCK_SIZE>>>(input, output); }

__global__ void sequential_cuda_kernel(const float *__restrict__ input, float *__restrict__ output) {
    float reg[N / BLOCK_SIZE];

#pragma unroll
    for (int s = 0; s < N; s += BLOCK_SIZE) {
        reg[s / BLOCK_SIZE] = input[s + threadIdx.x];
    }

#pragma unroll
    for (int i = 0; i < N / BLOCK_SIZE; i++) {
        reg[i] = logf(expf(cosf(sinf(reg[i]))));
    }

#pragma unroll
    for (int s = 0; s < N; s += BLOCK_SIZE) {
        output[s + threadIdx.x] = reg[s / BLOCK_SIZE];
    }
}

void sequential_cuda(const float *input, float *output) { sequential_cuda_kernel<<<1, BLOCK_SIZE>>>(input, output); }

int main() {
    float *h_input, *h_output_int, *h_output_seq;

    CHECK_CUDA(hipHostMalloc(&h_input, N * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_output_int, N * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_output_seq, N * sizeof(float)));

    float *d_input, *d_output_int, *d_output_seq;

    CHECK_CUDA(hipMalloc(&d_input, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_int, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_seq, N * sizeof(float)));

    for (int i = 0; i < N; i++) {
        h_input[i] = uniform();
    }

    CHECK_CUDA(hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice));

    interleaved_cuda(d_input, d_output_int);
    CHECK_CUDA(hipMemcpy(h_output_int, d_output_int, N * sizeof(float), hipMemcpyDeviceToHost));

    sequential_cuda(d_input, d_output_seq);
    CHECK_CUDA(hipMemcpy(h_output_seq, d_output_seq, N * sizeof(float), hipMemcpyDeviceToHost));

    // check correctness
    check_is_close(h_output_int, h_output_seq, N);

    const float int_elapsed = timeit([=] { interleaved_cuda(d_input, d_output_int); }, 100, 10000);
    const float seq_elapsed = timeit([=] { sequential_cuda(d_input, d_output_seq); }, 100, 10000);

    printf("interleaved: %.3f us\n", int_elapsed * 1e6f);
    printf("sequential:  %.3f us\n", seq_elapsed * 1e6f);

    CHECK_CUDA(hipHostFree(h_input));
    CHECK_CUDA(hipHostFree(h_output_int));
    CHECK_CUDA(hipHostFree(h_output_seq));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output_int));
    CHECK_CUDA(hipFree(d_output_seq));

    return 0;
}