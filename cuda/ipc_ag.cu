#include "hip/hip_runtime.h"
/*
See https://github.com/NVIDIA/cuda-samples/blob/master/Samples/0_Introduction/simpleIPC/simpleIPC.cu
Usage: mpirun -np 8 bin/ipc
*/

#include "common.h"
#include <mpi.h>
#include <vector>

// From https://github.com/NVIDIA/TensorRT-LLM/blob/main/cpp/tensorrt_llm/kernels/customAllReduceKernels.cu
static inline __device__ void st_flag_release(uint32_t const &flag, uint32_t *flag_addr) {
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

static inline __device__ uint32_t ld_flag_acquire(uint32_t *flag_addr) {
    uint32_t flag;
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
    return flag;
}

__global__ void ipc_all_gather_kernel(const int *__restrict__ input, int **__restrict__ peers_output,
                                      uint32_t **__restrict__ peers_flag, uint32_t flag_value, int rank, int world_size,
                                      int N) {
    // flag: [world_size, world_size]

    uint32_t *local_flag = peers_flag[rank];

    const int peer_rank = (blockIdx.y + rank) % world_size;
    int *peer_output = peers_output[peer_rank] + rank * N;
    for (int i = 4 * (blockIdx.x * blockDim.x + threadIdx.x); i < N; i += 4 * gridDim.x * blockDim.x) {
        *(float4 *)&peer_output[i] = *(float4 *)&input[i];
    }

    __shared__ int prev_flag;
    if (threadIdx.x == 0) {
        prev_flag = atomicAdd(local_flag + rank, 1);
    }
    __syncthreads();

    if (prev_flag == flag_value - 1) {
        if (threadIdx.x < world_size) {
            st_flag_release(flag_value, peers_flag[threadIdx.x] + rank);

            while (ld_flag_acquire(local_flag + threadIdx.x) != flag_value) {
            }
        }
    }
}

void ipc_all_gather_cuda(const int *input, int **peers_output, uint32_t **peers_flag, uint32_t *flag, int run_count,
                         int rank, int world_size, int N) {
    constexpr int block_size = 128;
    const dim3 grid_size((N / 4 + block_size - 1) / block_size, world_size);
    uint32_t flag_value = (run_count + 1) * grid_size.x * grid_size.y;
    ipc_all_gather_kernel<<<grid_size, block_size>>>(input, peers_output, peers_flag, flag_value, rank, world_size, N);
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);

    int world_size, rank;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    printf("[rank %d] initialized world size %d\n", rank, world_size);

    CHECK_CUDA(hipSetDevice(rank));

    const int N = 2 * 1024 * 1024;

    int *h_output;
    CHECK_CUDA(hipHostMalloc(&h_output, world_size * N * sizeof(int)));

    int *d_input;
    CHECK_CUDA(hipMalloc(&d_input, N * sizeof(int)));
    CHECK_CUDA(hipMemset(d_input, rank, N * sizeof(int)));

    std::vector<int *> d_output_h_vec(world_size);
    CHECK_CUDA(hipMalloc(&d_output_h_vec[rank], world_size * N * sizeof(int)));

    std::vector<uint32_t *> d_flag_h_vec(world_size);
    CHECK_CUDA(hipMalloc(&d_flag_h_vec[rank], world_size * sizeof(uint32_t)));
    CHECK_CUDA(hipMemset(d_flag_h_vec[rank], 0, world_size * sizeof(uint32_t)));

    // ipc mem
    std::vector<hipIpcMemHandle_t> mem_handles(world_size);
    CHECK_CUDA(hipIpcGetMemHandle(mem_handles.data() + rank, d_output_h_vec[rank]));
    MPI_Allgather(mem_handles.data() + rank, sizeof(hipIpcMemHandle_t), MPI_BYTE, mem_handles.data(),
                  sizeof(hipIpcMemHandle_t), MPI_BYTE, MPI_COMM_WORLD);
    for (int i = 0; i < world_size; i++) {
        if (i != rank) {
            CHECK_CUDA(
                hipIpcOpenMemHandle((void **)&d_output_h_vec[i], mem_handles[i], hipIpcMemLazyEnablePeerAccess));
        }
    }
    int **d_output_d_vec;
    CHECK_CUDA(hipMalloc(&d_output_d_vec, world_size * sizeof(void *)));
    CHECK_CUDA(
        hipMemcpyAsync(d_output_d_vec, d_output_h_vec.data(), world_size * sizeof(void *), hipMemcpyHostToDevice));

    CHECK_CUDA(hipIpcGetMemHandle(mem_handles.data() + rank, d_flag_h_vec[rank]));
    MPI_Allgather(mem_handles.data() + rank, sizeof(hipIpcMemHandle_t), MPI_BYTE, mem_handles.data(),
                  sizeof(hipIpcMemHandle_t), MPI_BYTE, MPI_COMM_WORLD);
    for (int i = 0; i < world_size; i++) {
        if (i != rank) {
            CHECK_CUDA(hipIpcOpenMemHandle((void **)&d_flag_h_vec[i], mem_handles[i], hipIpcMemLazyEnablePeerAccess));
        }
    }
    uint32_t **d_flag_d_vec;
    CHECK_CUDA(hipMalloc(&d_flag_d_vec, world_size * sizeof(void *)));
    CHECK_CUDA(hipMemcpyAsync(d_flag_d_vec, d_flag_h_vec.data(), world_size * sizeof(void *), hipMemcpyHostToDevice));

    // ipc events
    // std::vector<hipEvent_t> events(world_size);
    // std::vector<hipIpcEventHandle_t> event_handles(world_size);
    // CHECK_CUDA(hipEventCreate(&events[rank], hipEventDisableTiming | hipEventInterprocess));
    // CHECK_CUDA(hipIpcGetEventHandle(&event_handles[rank], events[rank]));
    // MPI_Allgather(event_handles.data() + rank, sizeof(hipIpcEventHandle_t), MPI_BYTE, event_handles.data(),
    //               sizeof(hipIpcEventHandle_t), MPI_BYTE, MPI_COMM_WORLD);
    // for (int i = 0; i < world_size; i++) {
    //     if (i != rank) {
    //         CHECK_CUDA(hipIpcOpenEventHandle(&events[i], event_handles[i]));
    //     }
    // }

    int run_count = 0;

    // run & check
    ipc_all_gather_cuda(d_input, d_output_d_vec, d_flag_d_vec, d_flag_h_vec[rank], run_count++, rank, world_size, N);
    CHECK_CUDA(hipMemcpy(h_output, d_output_h_vec[rank], world_size * N * sizeof(int), hipMemcpyDeviceToHost));

    int *h_output_ref;
    CHECK_CUDA(hipHostMalloc(&h_output_ref, world_size * N * sizeof(int)));
    for (int i = 0; i < world_size; i++) {
        memset(h_output_ref + i * N, i, N * sizeof(int));
    }
    CHECK(memcmp(h_output, h_output_ref, world_size * N * sizeof(int)) == 0);

    // benchmark
    const float elapsed = timeit(
        [&] {
            ipc_all_gather_cuda(d_input, d_output_d_vec, d_flag_d_vec, d_flag_h_vec[rank], run_count++, rank,
                                world_size, N);
        },
        10, 1000);
    const float bus_bandwidth = (world_size - 1) * N * sizeof(int) / 1e9f / elapsed;
    printf("[rank %d] [cuda] elapsed %.3f us, (uni-directional) bus_bandwidth %.3f GB/s\n", rank, elapsed * 1e6f,
           bus_bandwidth);

    MPI_Barrier(MPI_COMM_WORLD);

    // clean up
    for (int i = 0; i < world_size; i++) {
        if (i != rank) {
            CHECK_CUDA(hipIpcCloseMemHandle(d_output_h_vec[i]));
        }
    }

    CHECK_CUDA(hipHostFree(h_output));
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output_h_vec[rank]));
    CHECK_CUDA(hipFree(d_flag_h_vec[rank]));
    CHECK_CUDA(hipFree(d_output_d_vec));
    CHECK_CUDA(hipFree(d_flag_d_vec));
    CHECK_CUDA(hipHostFree(h_output_ref));

    MPI_Finalize();

    return 0;
}