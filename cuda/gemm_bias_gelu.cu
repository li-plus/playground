#include "hip/hip_runtime.h"
/*
cublasLt example: https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtSgemm/sample_cublasLt_LtSgemm.cu
PyTorch code: https://github.com/pytorch/pytorch/blob/main/aten/src/ATen/cuda/CUDABlas.cpp
*/

#include "common.h"
#include <hipblaslt.h>

void gemm_bias_gelu_cublaslt(hipblasLtHandle_t handle, const float *input, const float *weight, const float *bias,
                             float *output, int M, int N, int K, void *workspace, size_t workspace_size) {
    const float *A = weight; // [K, N] in row major -> [N, K] in col major
    const float *B = input;  // [M, K] in row major -> [K, M] in col major
    float *C = output;       // [M, N] in row major -> [N, M] in col major = OP_N(A) @ OP_N(B)

    const float alpha = 1.f;
    const float beta = 0.f;

    hipblasLtMatmulDesc_t matmul_desc;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmul_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    hipblasOperation_t trans_A = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans_A, sizeof(trans_A)));
    hipblasOperation_t trans_B = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &trans_B, sizeof(trans_B)));

    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_GELU_BIAS;
    CHECK_CUBLAS(
        hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmul_desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    hipblasLtMatrixLayout_t A_desc, B_desc, C_desc;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_32F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&B_desc, HIP_R_32F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&C_desc, HIP_R_32F, N, M, N));

    hipblasLtMatmulPreference_t preference;
    CHECK_CUBLAS(hipblasLtMatmulPreferenceCreate(&preference));
    CHECK_CUBLAS(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                                      &workspace_size, sizeof(workspace_size)));

    hipblasLtMatmulHeuristicResult_t heuristic_result{};
    int returned_result = 0;
    CHECK_CUBLAS(hipblasLtMatmulAlgoGetHeuristic(handle, matmul_desc, A_desc, B_desc, C_desc, C_desc, preference, 1,
                                                &heuristic_result, &returned_result));
    if (returned_result == 0) {
        CHECK_CUBLAS(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    CHECK_CUBLAS(hipblasLtMatmul(handle, matmul_desc, &alpha, A, A_desc, B, B_desc, &beta, C, C_desc, output, C_desc,
                                &heuristic_result.algo, workspace, workspace_size, 0));

    CHECK_CUBLAS(hipblasLtMatmulPreferenceDestroy(preference));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(A_desc));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(B_desc));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(C_desc));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(matmul_desc));
}

__device__ __forceinline__ float gelu(float x) {
    return 0.5f * x * (1.f + std::tanh(0.7978845608028654f * (x + 0.044715f * x * x * x)));
}

__global__ void bias_gelu_inplace_kernel(float *input, const float *bias, int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    input[idx] = gelu(input[idx] + bias[idx % N]);
}

hipError_t bias_gelu_inplace_cuda(float *input, const float *bias, int M, int N) {
    constexpr int block_size = 128;
    const int grid_size = M * N / block_size;
    bias_gelu_inplace_kernel<<<grid_size, block_size>>>(input, bias, N);
    return hipGetLastError();
}

void gemm_bias_gelu_cublas(hipblasHandle_t handle, const float *input, const float *weight, const float *bias,
                           float *output, int M, int N, int K) {
    const float *A = weight;
    const float *B = input;
    float *C = output;
    const float alpha = 1;
    const float beta = 0;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, A, N, B, K, &beta, C, N));
    CHECK_CUDA(bias_gelu_inplace_cuda(output, bias, M, N));
}

void benchmark(int M, int N, int K) {
    float *h_input, *h_weight, *h_bias, *h_output_expect, *h_output_actual;
    CHECK_CUDA(hipHostMalloc(&h_input, M * K * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_weight, K * N * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_bias, N * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_output_expect, M * N * sizeof(float)));
    CHECK_CUDA(hipHostMalloc(&h_output_actual, M * N * sizeof(float)));

    float *d_input, *d_weight, *d_bias, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_weight, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_bias, N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, M * N * sizeof(float)));

    void *workspace;
    const size_t workspace_size = 1024 * 1024;
    CHECK_CUDA(hipMalloc(&workspace, workspace_size));

    for (int i = 0; i < M * K; i++) {
        h_input[i] = uniform(-0.5, 0.5);
    }
    for (int i = 0; i < K * N; i++) {
        h_weight[i] = uniform(-0.5, 0.5);
    }
    CHECK_CUDA(hipMemcpyAsync(d_input, h_input, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpyAsync(d_weight, h_weight, K * N * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    hipblasLtHandle_t cublaslt_handle;
    CHECK_CUBLAS(hipblasLtCreate(&cublaslt_handle));

    {
        CHECK_CUDA(hipMemsetAsync(d_output, 0, M * N * sizeof(float)));
        gemm_bias_gelu_cublas(cublas_handle, d_input, d_weight, d_bias, d_output, M, N, K);
        CHECK_CUDA(hipMemcpy(h_output_actual, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));
    }
    {
        CHECK_CUDA(hipMemsetAsync(d_output, 0, M * N * sizeof(float)));
        gemm_bias_gelu_cublaslt(cublaslt_handle, d_input, d_weight, d_bias, d_output, M, N, K, workspace,
                                workspace_size);
        CHECK_CUDA(hipMemcpy(h_output_expect, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost));
    }
    check_is_close(h_output_actual, h_output_expect, M * N, 1e-4, 1e-4);

    const float tflops = (2.f * M * N * K) / 1e12;
    const float gbytes = (M * K + K * N + M * N) / 1e9;
    {
        const float elapsed = timeit(
            [&] { gemm_bias_gelu_cublas(cublas_handle, d_input, d_weight, d_bias, d_output, M, N, K); }, 10, 100);
        printf("[cublas] elapsed: %.3f us, throughput: %.3f TFLOPS, bandwidth: %.3f GB/s\n", elapsed * 1e6f,
               tflops / elapsed, gbytes / elapsed);
    }
    {
        const float elapsed = timeit(
            [&] {
                gemm_bias_gelu_cublaslt(cublaslt_handle, d_input, d_weight, d_bias, d_output, M, N, K, workspace,
                                        workspace_size);
            },
            10, 100);
        printf("[cublasLt] elapsed: %.3f us, throughput: %.3f TFLOPS, bandwidth: %.3f GB/s\n", elapsed * 1e6f,
               tflops / elapsed, gbytes / elapsed);
    }

    CHECK_CUDA(hipFree(workspace));

    CHECK_CUDA(hipHostFree(h_input));
    CHECK_CUDA(hipHostFree(h_weight));
    CHECK_CUDA(hipHostFree(h_bias));
    CHECK_CUDA(hipHostFree(h_output_expect));
    CHECK_CUDA(hipHostFree(h_output_actual));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_weight));
    CHECK_CUDA(hipFree(d_bias));
    CHECK_CUDA(hipFree(d_output));
}

int main(int argc, char **argv) {
    int M = 4096, N = 1024, K = 2048;
    benchmark(M, N, K);
    return 0;
}
