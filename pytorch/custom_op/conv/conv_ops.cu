#include <ATen/cuda/Exceptions.h>
#include <ATen/cudnn/Handle.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <torch/extension.h>

#define CHECK_CUDNN(status) AT_CUDNN_CHECK(status, " at " __FILE__ ":", __LINE__)

torch::Tensor conv2d(torch::Tensor input, torch::Tensor weight, std::optional<torch::Tensor> bias,
                     const std::vector<int> &stride, const std::vector<int> &padding,
                     const std::vector<int> &dilation) {
    TORCH_CHECK(stride.size() == 2);
    TORCH_CHECK(padding.size() == 2);
    TORCH_CHECK(dilation.size() == 2);

    hipdnnHandle_t handle = at::native::getCudnnHandle();

    const int N = input.size(0);
    const int IC = input.size(1);
    const int IH = input.size(2);
    const int IW = input.size(3);

    const int OC = weight.size(0);
    TORCH_CHECK(IC == weight.size(1));
    const int KH = weight.size(2);
    const int KW = weight.size(3);

    int PH = padding.at(0), PW = padding.at(1);
    int SH = stride.at(0), SW = stride.at(1);
    int DH = dilation.at(0), DW = dilation.at(1);

    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(
        hipdnnSetConvolution2dDescriptor(conv_desc, PH, PW, SH, SW, DH, DW, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    hipdnnTensorDescriptor_t input_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, IC, IH, IW));

    hipdnnFilterDescriptor_t weight_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&weight_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OC, IC, KH, KW));

    int ON, OC_COMP, OH, OW;
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, weight_desc, &ON, &OC_COMP, &OH, &OW));
    TORCH_CHECK(ON == N && OC_COMP == OC);

    hipdnnTensorDescriptor_t output_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, OC, OH, OW));

    torch::Tensor output = torch::empty({N, OC, OH, OW}, input.options());

    const float *input_ptr = input.const_data_ptr<float>();
    const float *weight_ptr = weight.const_data_ptr<float>();
    float *output_ptr = output.mutable_data_ptr<float>();

    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, weight_desc, conv_desc, output_desc, algo,
                                                        &workspace_size));

    auto workspace = c10::cuda::CUDACachingAllocator::get()->allocate(workspace_size);

    const float alpha = 1.f;
    const float beta = 0.f;
    if (bias) {
        TORCH_CHECK(bias->ndimension() == 1 && bias->numel() == OC);
        hipdnnTensorDescriptor_t bias_desc;
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&bias_desc));
        CHECK_CUDNN(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, OC, 1, 1));
        const float *bias_ptr = bias->const_data_ptr<float>();

        hipdnnActivationDescriptor_t act_desc;
        CHECK_CUDNN(hipdnnCreateActivationDescriptor(&act_desc));
        CHECK_CUDNN(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

        CHECK_CUDNN(cudnnConvolutionBiasActivationForward(
            handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc, algo, workspace.get(),
            workspace_size, &beta, output_desc, output_ptr, bias_desc, bias_ptr, act_desc, output_desc, output_ptr));

        CHECK_CUDNN(hipdnnDestroyActivationDescriptor(act_desc));
        CHECK_CUDNN(hipdnnDestroyTensorDescriptor(bias_desc));
    } else {
        CHECK_CUDNN(hipdnnConvolutionForward(handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc,
                                            algo, workspace.get(), workspace_size, &beta, output_desc, output_ptr));
    }

    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(weight_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));

    return output;
}

torch::Tensor convnd(torch::Tensor input, torch::Tensor weight, std::optional<torch::Tensor> bias,
                     const std::vector<int> &stride, const std::vector<int> &padding,
                     const std::vector<int> &dilation) {
    TORCH_CHECK(input.ndimension() >= 3);
    hipdnnHandle_t handle = at::native::getCudnnHandle();

    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolutionNdDescriptor(conv_desc, stride.size(), padding.data(), stride.data(),
                                                dilation.data(), HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    std::vector<int> input_dims(input.sizes().begin(), input.sizes().end());
    std::vector<int> input_strides(input.strides().begin(), input.strides().end());
    hipdnnTensorDescriptor_t input_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(input_desc, HIPDNN_DATA_FLOAT, input.ndimension(), input_dims.data(),
                                           input_strides.data()));

    std::vector<int> weight_dims(weight.sizes().begin(), weight.sizes().end());
    hipdnnFilterDescriptor_t weight_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&weight_desc));
    CHECK_CUDNN(hipdnnSetFilterNdDescriptor(weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, weight.ndimension(),
                                           weight_dims.data()));

    std::vector<int> output_dims(input.ndimension());

    CHECK_CUDNN(cudnnGetConvolutionNdForwardOutputDim(conv_desc, input_desc, weight_desc, output_dims.size(),
                                                      output_dims.data()));
    TORCH_CHECK(output_dims.at(0) == input_dims.at(0) && output_dims.at(1) == weight_dims.at(0));

    torch::Tensor output = torch::empty(std::vector<long>(output_dims.begin(), output_dims.end()), input.options());
    std::vector<int> output_strides(output.strides().begin(), output.strides().end());

    hipdnnTensorDescriptor_t output_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(output_desc, HIPDNN_DATA_FLOAT, output.ndimension(), output_dims.data(),
                                           output_strides.data()));

    const float *input_ptr = input.const_data_ptr<float>();
    const float *weight_ptr = weight.const_data_ptr<float>();
    float *output_ptr = output.mutable_data_ptr<float>();

    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, weight_desc, conv_desc, output_desc, algo,
                                                        &workspace_size));

    auto workspace = c10::cuda::CUDACachingAllocator::get()->allocate(workspace_size);

    const float alpha = 1.f;
    const float beta = 0.f;
    if (bias) {
        TORCH_CHECK(bias->ndimension() == 1 && bias->numel() == weight.size(0));
        std::vector<int> bias_dims(output.ndimension(), 1);
        bias_dims.at(1) = bias->numel();
        std::vector<int> bias_strides(output.ndimension(), bias->stride(0));
        hipdnnTensorDescriptor_t bias_desc;
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&bias_desc));
        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(bias_desc, HIPDNN_DATA_FLOAT, bias_dims.size(), bias_dims.data(),
                                               bias_strides.data()));
        const float *bias_ptr = bias->const_data_ptr<float>();

        hipdnnActivationDescriptor_t act_desc;
        CHECK_CUDNN(hipdnnCreateActivationDescriptor(&act_desc));
        CHECK_CUDNN(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

        CHECK_CUDNN(cudnnConvolutionBiasActivationForward(
            handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc, algo, workspace.get(),
            workspace_size, &beta, output_desc, output_ptr, bias_desc, bias_ptr, act_desc, output_desc, output_ptr));

        CHECK_CUDNN(hipdnnDestroyActivationDescriptor(act_desc));
        CHECK_CUDNN(hipdnnDestroyTensorDescriptor(bias_desc));
    } else {
        CHECK_CUDNN(hipdnnConvolutionForward(handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc,
                                            algo, workspace.get(), workspace_size, &beta, output_desc, output_ptr));
    }

    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(weight_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));

    return output;
}

torch::Tensor conv1d(torch::Tensor input, torch::Tensor weight, std::optional<torch::Tensor> bias,
                     const std::vector<int> &stride, const std::vector<int> &padding,
                     const std::vector<int> &dilation) {
    torch::Tensor output = ::conv2d(input.unsqueeze(-2), weight.unsqueeze(-2), bias, {1, stride.at(0)},
                                    {0, padding.at(0)}, {1, dilation.at(0)});
    return output.squeeze(-2);
}

inline hipdnnTensorDescriptor_t create_tensor_descriptor(torch::Tensor tensor) {
    std::vector<int> tensor_dims(tensor.sizes().begin(), tensor.sizes().end());
    std::vector<int> tensor_strides(tensor.strides().begin(), tensor.strides().end());
    hipdnnTensorDescriptor_t tensor_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&tensor_desc));
    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(tensor_desc, HIPDNN_DATA_FLOAT, tensor.ndimension(), tensor_dims.data(),
                                           tensor_strides.data()));
    return tensor_desc;
}

inline void destroy_tensor_descriptor(hipdnnTensorDescriptor_t tensor_desc) {
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(tensor_desc));
}

inline hipdnnFilterDescriptor_t create_filter_descriptor(torch::Tensor weight) {
    std::vector<int> weight_dims(weight.sizes().begin(), weight.sizes().end());
    std::vector<int> weight_strides(weight.strides().begin(), weight.strides().end());
    hipdnnFilterDescriptor_t weight_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&weight_desc));
    CHECK_CUDNN(hipdnnSetFilterNdDescriptor(weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, weight_dims.size(),
                                           weight_dims.data()));
    return weight_desc;
}

inline void destroy_filter_descriptor(hipdnnFilterDescriptor_t filter_desc) {
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(filter_desc));
}

inline hipdnnConvolutionDescriptor_t create_convolution_descriptor(const std::vector<int> &stride,
                                                                  const std::vector<int> &padding,
                                                                  const std::vector<int> &dilation) {
    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolutionNdDescriptor(conv_desc, padding.size(), padding.data(), stride.data(),
                                                dilation.data(), HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    return conv_desc;
}

inline void destroy_convolution_descriptor(hipdnnConvolutionDescriptor_t conv_desc) {
    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
}

torch::Tensor conv_backward_input(torch::Tensor grad_output, torch::Tensor input, torch::Tensor weight,
                                  const std::vector<int> &stride, const std::vector<int> &padding,
                                  const std::vector<int> &dilation) {
    torch::Tensor grad_input = torch::empty_like(input);

    hipdnnHandle_t handle = at::native::getCudnnHandle();

    hipdnnConvolutionDescriptor_t conv_desc = create_convolution_descriptor(stride, padding, dilation);

    hipdnnTensorDescriptor_t grad_output_desc = create_tensor_descriptor(grad_output);
    hipdnnFilterDescriptor_t weight_desc = create_filter_descriptor(weight);
    hipdnnTensorDescriptor_t grad_input_desc = create_tensor_descriptor(grad_input);

    hipdnnConvolutionBwdDataAlgo_t algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;

    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(handle, weight_desc, grad_output_desc, conv_desc,
                                                             grad_input_desc, algo, &workspace_size));
    auto workspace = c10::cuda::CUDACachingAllocator::get()->allocate(workspace_size);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    CHECK_CUDNN(hipdnnConvolutionBackwardData(handle, &alpha, weight_desc, weight.const_data_ptr<float>(),
                                             grad_output_desc, grad_output.const_data_ptr<float>(), conv_desc, algo,
                                             workspace.get(), workspace_size, &beta, grad_input_desc,
                                             grad_input.mutable_data_ptr<float>()));

    destroy_convolution_descriptor(conv_desc);

    destroy_tensor_descriptor(grad_output_desc);
    destroy_filter_descriptor(weight_desc);
    destroy_tensor_descriptor(grad_input_desc);

    return grad_input;
}

torch::Tensor conv_backward_weight(torch::Tensor grad_output, torch::Tensor input, torch::Tensor weight,
                                   const std::vector<int> &stride, const std::vector<int> &padding,
                                   const std::vector<int> &dilation) {
    torch::Tensor grad_weight = torch::empty_like(weight);

    hipdnnHandle_t handle = at::native::getCudnnHandle();

    hipdnnConvolutionDescriptor_t conv_desc = create_convolution_descriptor(stride, padding, dilation);

    hipdnnTensorDescriptor_t grad_output_desc = create_tensor_descriptor(grad_output);
    hipdnnTensorDescriptor_t input_desc = create_tensor_descriptor(input);
    hipdnnFilterDescriptor_t grad_weight_desc = create_filter_descriptor(grad_weight);

    hipdnnConvolutionBwdFilterAlgo_t algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;

    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle, input_desc, grad_output_desc, conv_desc,
                                                               grad_weight_desc, algo, &workspace_size));
    auto workspace = c10::cuda::CUDACachingAllocator::get()->allocate(workspace_size);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    CHECK_CUDNN(hipdnnConvolutionBackwardFilter(handle, &alpha, input_desc, input.const_data_ptr<float>(),
                                               grad_output_desc, grad_output.const_data_ptr<float>(), conv_desc, algo,
                                               workspace.get(), workspace_size, &beta, grad_weight_desc,
                                               grad_weight.mutable_data_ptr<float>()));

    destroy_convolution_descriptor(conv_desc);

    destroy_tensor_descriptor(grad_output_desc);
    destroy_tensor_descriptor(input_desc);
    destroy_filter_descriptor(grad_weight_desc);

    return grad_weight;
}

torch::Tensor conv_backward_bias(torch::Tensor grad_output, torch::Tensor input, torch::Tensor weight,
                                 const std::vector<int> &stride, const std::vector<int> &padding,
                                 const std::vector<int> &dilation) {
    std::vector<long> bias_dims(weight.ndimension(), 1);
    bias_dims.at(1) = weight.size(0); // set channel
    torch::Tensor grad_bias = torch::empty(bias_dims, weight.options());

    hipdnnHandle_t handle = at::native::getCudnnHandle();

    hipdnnConvolutionDescriptor_t conv_desc = create_convolution_descriptor(stride, padding, dilation);

    hipdnnTensorDescriptor_t grad_output_desc = create_tensor_descriptor(grad_output);
    hipdnnFilterDescriptor_t weight_desc = create_filter_descriptor(weight);
    hipdnnTensorDescriptor_t grad_bias_desc = create_tensor_descriptor(grad_bias);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    CHECK_CUDNN(hipdnnConvolutionBackwardBias(handle, &alpha, grad_output_desc, grad_output.const_data_ptr<float>(),
                                             &beta, grad_bias_desc, grad_bias.mutable_data_ptr<float>()));

    destroy_convolution_descriptor(conv_desc);

    destroy_tensor_descriptor(grad_output_desc);
    destroy_filter_descriptor(weight_desc);

    return grad_bias.view({grad_bias.size(1)});
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
conv_backward(torch::Tensor grad_output, torch::Tensor input, torch::Tensor weight, const std::vector<int> &stride,
              const std::vector<int> &padding, const std::vector<int> &dilation, std::array<bool, 3> output_mask) {
    torch::Tensor grad_input;
    if (output_mask[0]) {
        grad_input = conv_backward_input(grad_output, input, weight, stride, padding, dilation);
    }

    torch::Tensor grad_weight;
    if (output_mask[1]) {
        grad_weight = conv_backward_weight(grad_output, input, weight, stride, padding, dilation);
    }

    torch::Tensor grad_bias;
    if (output_mask[2]) {
        grad_bias = conv_backward_bias(grad_output, input, weight, stride, padding, dilation);
    }

    return std::make_tuple(grad_input, grad_weight, grad_bias);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
conv1d_backward(torch::Tensor grad_output, torch::Tensor input, torch::Tensor weight, const std::vector<int> &stride,
                const std::vector<int> &padding, const std::vector<int> &dilation, std::array<bool, 3> output_mask) {
    auto [grad_input, grad_weight, grad_bias] =
        conv_backward(grad_output.unsqueeze(-2), input.unsqueeze(-2), weight.unsqueeze(-2), {1, stride.at(0)},
                      {0, padding.at(0)}, {1, dilation.at(0)}, output_mask);

    if (output_mask[0]) {
        grad_input.squeeze_(-2);
    }
    if (output_mask[1]) {
        grad_weight.squeeze_(-2);
    }

    return std::make_tuple(grad_input, grad_weight, grad_bias);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("conv3d", &convnd, "conv3d using cudnn");
    m.def("conv2d", &conv2d, "conv2d using cudnn");
    m.def("conv1d", &conv1d, "conv1d using cudnn");
    m.def("conv1d_backward", &conv1d_backward, "conv1d_backward using cudnn");
    m.def("conv2d_backward", &conv_backward, "conv2d_backward using cudnn");
    m.def("conv3d_backward", &conv_backward, "conv3d_backward using cudnn");
}
