#include <ATen/cuda/Exceptions.h>
#include <ATen/cudnn/Handle.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <torch/extension.h>

#define CHECK_CUDNN(status) AT_CUDNN_CHECK(status, " at " __FILE__ ":", __LINE__)

torch::Tensor conv2d(torch::Tensor input, torch::Tensor weight, std::optional<torch::Tensor> bias,
                     const std::vector<int> &stride, const std::vector<int> &padding,
                     const std::vector<int> &dilation) {
    TORCH_CHECK(stride.size() == 2);
    TORCH_CHECK(padding.size() == 2);
    TORCH_CHECK(dilation.size() == 2);

    hipdnnHandle_t handle = at::native::getCudnnHandle();

    const int N = input.size(0);
    const int IC = input.size(1);
    const int IH = input.size(2);
    const int IW = input.size(3);

    const int OC = weight.size(0);
    TORCH_CHECK(IC == weight.size(1));
    const int KH = weight.size(2);
    const int KW = weight.size(3);

    int PH = padding.at(0), PW = padding.at(1);
    int SH = stride.at(0), SW = stride.at(1);
    int DH = dilation.at(0), DW = dilation.at(1);

    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(
        hipdnnSetConvolution2dDescriptor(conv_desc, PH, PW, SH, SW, DH, DW, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    hipdnnTensorDescriptor_t input_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, IC, IH, IW));

    hipdnnFilterDescriptor_t weight_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&weight_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OC, IC, KH, KW));

    int ON, OC_COMP, OH, OW;
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, weight_desc, &ON, &OC_COMP, &OH, &OW));
    TORCH_CHECK(ON == N && OC_COMP == OC);

    hipdnnTensorDescriptor_t output_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, OC, OH, OW));

    torch::Tensor output = torch::empty({N, OC, OH, OW}, input.options());

    const float *input_ptr = input.const_data_ptr<float>();
    const float *weight_ptr = weight.const_data_ptr<float>();
    float *output_ptr = output.mutable_data_ptr<float>();

    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, weight_desc, conv_desc, output_desc, algo,
                                                        &workspace_size));

    auto &allocator = *c10::cuda::CUDACachingAllocator::get();
    auto workspace = allocator.allocate(workspace_size);

    const float alpha = 1.f;
    const float beta = 0.f;
    if (bias) {
        TORCH_CHECK(bias->ndimension() == 1 && bias->numel() == OC);
        hipdnnTensorDescriptor_t bias_desc;
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&bias_desc));
        CHECK_CUDNN(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, OC, 1, 1));
        const float *bias_ptr = bias->const_data_ptr<float>();

        hipdnnActivationDescriptor_t act_desc;
        CHECK_CUDNN(hipdnnCreateActivationDescriptor(&act_desc));
        CHECK_CUDNN(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

        CHECK_CUDNN(cudnnConvolutionBiasActivationForward(
            handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc, algo, workspace.get(),
            workspace_size, &beta, output_desc, output_ptr, bias_desc, bias_ptr, act_desc, output_desc, output_ptr));

        CHECK_CUDNN(hipdnnDestroyActivationDescriptor(act_desc));
        CHECK_CUDNN(hipdnnDestroyTensorDescriptor(bias_desc));
    } else {
        CHECK_CUDNN(hipdnnConvolutionForward(handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc,
                                            algo, workspace.get(), workspace_size, &beta, output_desc, output_ptr));
    }

    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(weight_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));

    return output;
}

torch::Tensor convnd(torch::Tensor input, torch::Tensor weight, std::optional<torch::Tensor> bias,
                     const std::vector<int> &stride, const std::vector<int> &padding,
                     const std::vector<int> &dilation) {
    TORCH_CHECK(input.ndimension() >= 3);
    hipdnnHandle_t handle = at::native::getCudnnHandle();

    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolutionNdDescriptor(conv_desc, stride.size(), padding.data(), stride.data(),
                                                dilation.data(), HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    std::vector<int> input_dims(input.sizes().begin(), input.sizes().end());
    std::vector<int> input_strides(input.strides().begin(), input.strides().end());
    hipdnnTensorDescriptor_t input_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(input_desc, HIPDNN_DATA_FLOAT, input.ndimension(), input_dims.data(),
                                           input_strides.data()));

    std::vector<int> weight_dims(weight.sizes().begin(), weight.sizes().end());
    hipdnnFilterDescriptor_t weight_desc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&weight_desc));
    CHECK_CUDNN(hipdnnSetFilterNdDescriptor(weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, weight.ndimension(),
                                           weight_dims.data()));

    std::vector<int> output_dims(input.ndimension());

    CHECK_CUDNN(cudnnGetConvolutionNdForwardOutputDim(conv_desc, input_desc, weight_desc, output_dims.size(),
                                                      output_dims.data()));
    TORCH_CHECK(output_dims.at(0) == input_dims.at(0) && output_dims.at(1) == weight_dims.at(0));

    torch::Tensor output = torch::empty(std::vector<long>(output_dims.begin(), output_dims.end()), input.options());
    std::vector<int> output_strides(output.strides().begin(), output.strides().end());

    hipdnnTensorDescriptor_t output_desc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensorNdDescriptor(output_desc, HIPDNN_DATA_FLOAT, output.ndimension(), output_dims.data(),
                                           output_strides.data()));

    const float *input_ptr = input.const_data_ptr<float>();
    const float *weight_ptr = weight.const_data_ptr<float>();
    float *output_ptr = output.mutable_data_ptr<float>();

    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    size_t workspace_size;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, weight_desc, conv_desc, output_desc, algo,
                                                        &workspace_size));

    auto &allocator = *c10::cuda::CUDACachingAllocator::get();
    auto workspace = allocator.allocate(workspace_size);

    const float alpha = 1.f;
    const float beta = 0.f;
    if (bias) {
        TORCH_CHECK(bias->ndimension() == 1 && bias->numel() == weight.size(0));
        std::vector<int> bias_dims(output.ndimension(), 1);
        bias_dims.at(1) = bias->numel();
        std::vector<int> bias_strides(output.ndimension(), bias->stride(0));
        hipdnnTensorDescriptor_t bias_desc;
        CHECK_CUDNN(hipdnnCreateTensorDescriptor(&bias_desc));
        CHECK_CUDNN(hipdnnSetTensorNdDescriptor(bias_desc, HIPDNN_DATA_FLOAT, bias_dims.size(), bias_dims.data(),
                                               bias_strides.data()));
        const float *bias_ptr = bias->const_data_ptr<float>();

        hipdnnActivationDescriptor_t act_desc;
        CHECK_CUDNN(hipdnnCreateActivationDescriptor(&act_desc));
        CHECK_CUDNN(hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));

        CHECK_CUDNN(cudnnConvolutionBiasActivationForward(
            handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc, algo, workspace.get(),
            workspace_size, &beta, output_desc, output_ptr, bias_desc, bias_ptr, act_desc, output_desc, output_ptr));

        CHECK_CUDNN(hipdnnDestroyActivationDescriptor(act_desc));
        CHECK_CUDNN(hipdnnDestroyTensorDescriptor(bias_desc));
    } else {
        CHECK_CUDNN(hipdnnConvolutionForward(handle, &alpha, input_desc, input_ptr, weight_desc, weight_ptr, conv_desc,
                                            algo, workspace.get(), workspace_size, &beta, output_desc, output_ptr));
    }

    CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK_CUDNN(hipdnnDestroyFilterDescriptor(weight_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));

    return output;
}

torch::Tensor conv1d(torch::Tensor input, torch::Tensor weight, std::optional<torch::Tensor> bias,
                     const std::vector<int> &stride, const std::vector<int> &padding,
                     const std::vector<int> &dilation) {
    torch::Tensor output = ::conv2d(input.unsqueeze(-2), weight.unsqueeze(-2), bias, {1, stride.at(0)},
                                    {0, padding.at(0)}, {1, dilation.at(0)});
    return output.squeeze(-2);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("conv3d", &convnd, "conv3d using cudnn");
    m.def("conv2d", &conv2d, "conv2d using cudnn");
    m.def("conv1d", &conv1d, "conv1d using cudnn");
}
